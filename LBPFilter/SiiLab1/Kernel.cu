#include "hip/hip_runtime.h"
//Cuda file 
#include "hip/hip_runtime.h"  
#include "" 
#include <opencv2/core/core.hpp>  
#include <opencv2/highgui/highgui.hpp>  
#include <opencv2/imgproc/types_c.h>  
#include <opencv2/imgproc/imgproc.hpp>  



using namespace cv; 

__global__ void Kernel_LBPFilter(unsigned char *MatA, unsigned char *MatR, int rows, int cols);

__device__  int Mask(int pi, int po);

int iDivUp(int a, int b);

extern "C"	hipError_t ApplyLBPFilter(Mat *ptMatA, Mat *ptMatR)
{
	hipError_t status;

	//pointeurs des matrices 
	uchar *MatA, *MatR;

	//Dimension de la grid et des blocs 
	dim3 nbreThreadsParBlock(32, 32);
	dim3 nbreBloc(iDivUp(ptMatA->cols, 32), iDivUp(ptMatA->rows, 32));

	//Calculer l'espace n�cessaire dans la m�moire du gpu
	int memSize = ptMatA->rows * ptMatA->step1();

	//Allouer espace pour le gpu 
	hipMalloc((void **)&MatA, memSize);
	hipMalloc((void **)&MatR, memSize);

	status = hipGetLastError();
	if (status != hipSuccess)
	{
		goto Error;
	}
	//Envoyer matrice dans la m�moire du gpu 
	hipMemcpy(MatA, ptMatA->data, memSize, hipMemcpyHostToDevice);
	//Check status
	status = hipGetLastError();
	if (status != hipSuccess)
	{
		goto Error;
	}

	Kernel_LBPFilter <<<nbreBloc, nbreThreadsParBlock >>>(MatA, MatR, ptMatA->step1(), ptMatA->rows);

	//Check status
	status = hipGetLastError();
	if (status != hipSuccess)
	{
		goto Error;
	}
	//Wait the Kernel to be done
	hipDeviceSynchronize();
	//Retourner la matrice r�sultante 
	hipMemcpy(ptMatR->data, MatR, memSize, hipMemcpyDeviceToHost);
	//Check status
	status = hipGetLastError();
	if (status != hipSuccess)
	{
		goto Error;
	}
	//Lib�rer espace m�moire dans le gpu
	hipFree(MatA);
	hipFree(MatR);
	return status;
Error:
	hipFree(MatA);
	hipFree(MatR);
	return status;
}
 
__global__ void Kernel_LBPFilter(unsigned char *MatA, unsigned char *MatR, int rows, int cols)
{

	//X et Y dans la matrice 
	int ImgNumColonne = (blockIdx.x  * blockDim.x) + threadIdx.x;
	int ImgNumLigne = (blockIdx.y * blockDim.y) + threadIdx.y;

	//Ne depasse pas l'acc�s de la matrice
	if ((ImgNumColonne < (rows)-1) && (ImgNumLigne < (cols)-2))
	{
		
		//Total addition
		int total = 0;
		//Exposant
		int exp = 1;
		//Indice du Po
		int x5 = ((ImgNumLigne + 1) * rows) + ((ImgNumColonne)* 3) + 1;
		//Valeur du po (initial)
		int po = MatA[x5];

		//Pour chaque ligne(3)
		for (int iL = 1; iL <= 3; iL++) 
		{
			//Pour chaque colonne(3)
			for (int iC = 1; iC <= 3; iC++) 
			{
				//(Valeur de mon indice)
				int xpi = ((ImgNumLigne + iL - 1) * rows) + ((ImgNumColonne * 3) + (iC - 1));

				//Pour ne pas calculer le po
				if (xpi != x5) 
				{
					//Masque
					int mpi = Mask(MatA[xpi], po);

					mpi = mpi * exp;
					total = total + mpi;
					exp = exp * 2;
				}
				
			}
		}
		//Mettre r�sultat dans la matrice
		MatR[x5] = total;
	}

}

//Retourne 0 ou 1 (Pi = 0 si Pi<0 sinon =1)
__device__  int Mask(int pi, int po) 
{
	if (pi >= po) 
	{
		return 1;
	}
	else 
	{
		return 0;
	}
}

int iDivUp(int a, int b) // Round a / b to nearest higher integer value

{
	return (a % b != 0) ? (a / b + 1) : (a / b);
}


 
 